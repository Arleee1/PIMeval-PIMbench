// Test: C++ version of the stencil
// Copyright (c) 2025 University of Virginia
// This file is licensed under the MIT License.
// See the LICENSE file in the root of this repository for more details.

#include <iostream>
#include <vector>
#include <getopt.h>
#include <stdint.h>
#include <iomanip>
#include <cassert>
#include <type_traits>
#include <queue>
#include <random>
#include <limits>
#include <algorithm>
#include <list>
#include <cstring>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>
#if defined(_OPENMP)
#include <omp.h>
#endif

// Params ---------------------------------------------------------------------
typedef struct Params
{
  uint64_t gridWidth;
  uint64_t gridHeight;
  uint64_t stencilWidth;
  uint64_t stencilHeight;
  uint64_t numLeft;
  uint64_t numAbove;
  const char *configFile;
  const char *inputFile;
  bool shouldVerify;
} Params;

void usage()
{
  fprintf(stderr,
          "\nUsage:  ./stencil.out [options]"
          "\n"
          "\n    -x    grid width (default=2048 elements)"
          "\n    -y    grid height (default=2048 elements)"
          "\n    -w    horizontal stencil size (default=3)"
          "\n    -d    vertical stencil size (default=3)"
          "\n    -l    number of elements to the left of the output element for the stencil pattern, must be less than the horizontal stencil size (default=1)"
          "\n    -a    number of elements above the output element for the stencil pattern, must be less than the vertical stencil size (default=1)"
          "\n    -i    input file containing a 2d array (default=random)"
          "\n    -v    t = verifies PIM output with host output. (default=false)"
          "\n");
}

struct Params getInputParams(int argc, char **argv)
{
  struct Params p;
  p.gridWidth = 2048;
  p.gridHeight = 2048;
  p.stencilWidth = 3;
  p.stencilHeight = 3;
  p.numLeft = 1;
  p.numAbove = 1;
  p.inputFile = nullptr;
  p.shouldVerify = false;

  int opt;
  while ((opt = getopt(argc, argv, "h:x:y:w:d:l:a:i:v:")) >= 0)
  {
    switch (opt)
    {
    case 'h':
      usage();
      exit(0);
      break;
    case 'x':
      p.gridWidth = strtoull(optarg, NULL, 0);
      break;
    case 'y':
      p.gridHeight = strtoull(optarg, NULL, 0);
      break;
    case 'w':
      p.stencilWidth = strtoull(optarg, NULL, 0);
      break;
    case 'd':
      p.stencilHeight = strtoull(optarg, NULL, 0);
      break;
    case 'l':
      p.numLeft = strtoull(optarg, NULL, 0);
      break;
    case 'a':
      p.numAbove = strtoull(optarg, NULL, 0);
      break;
    case 'i':
      p.inputFile = optarg;
      break;
    case 'v':
      p.shouldVerify = (*optarg == 't');
      break;
    default:
      fprintf(stderr, "\nUnrecognized option!\n");
      usage();
      exit(0);
    }
  }
  return p;
}

inline __device__ uint64_t getIdxFromPos(const uint64_t x, const uint64_t y, const uint64_t width) {
  return y*width + x;
}

//! @brief  Computes a stencil average over a 2d array using CUDA. Grid cells who's stencil pattern lies partly outside of the input data range are undefined.
//! @param[in]  src  Device pointer to stencil input data
//! @param[out]  dst  Device pointer to stencil output data
//! @param[in]  toDivideBy  The number to divide the stencil sums by (Will be the number of cells in the stencil pattern for an average)
//! @param[in]  gridWidth  The width of the input and output grid
//! @param[in]  gridHeight  The height of the input and output grid
//! @param[in]  stencilWidth  The horizontal width of the stencil average rectangle
//! @param[in]  stencilHeight  The vertical height of the stencil average rectangle
//! @param[in]  numLeft  The number of elements to the left of the result grid element
//! @param[in]  numAbove  The number of elements to the right of the result grid element
template <typename StencilType>
__global__ void rectangleStencilAverage(
  const StencilType* src,
  StencilType* dst,
  const StencilType toDivideBy,
  const uint64_t gridWidth,
  const uint64_t gridHeight,
  const uint64_t stencilWidth,
  const uint64_t stencilHeight,
  const uint64_t numLeft,
  const uint64_t numRight,
  const uint64_t numAbove,
  const uint64_t numBelow
) {
  const uint64_t xPos = blockDim.x * blockIdx.x + threadIdx.x + numLeft;
	const uint64_t yPos = blockDim.y * blockIdx.y + threadIdx.y + numAbove;

  if((xPos + numRight) >= gridWidth || (yPos + numBelow) >= gridHeight) {
    return;
  }

  const uint64_t idx = getIdxFromPos(xPos, yPos, gridWidth);

  StencilType output = 0;
  for(uint64_t y=yPos-numAbove; y<=yPos+numBelow; ++y) {
    for(uint64_t x=xPos-numLeft; x<=xPos+numRight; ++x) {
      output += src[getIdxFromPos(x, y, gridWidth)];
    }
  }

  dst[idx] = output / toDivideBy;
}

//! @brief  Computes a stencil pattern over a 2d array
//! @param[in]  srcHost  The input stencil grid
//! @param[in]  dstHost  The resultant stencil grid
//! @param[in]  gridWidth  The width of the stencil grid
//! @param[in]  gridHeight  The height of the stencil grid
//! @param[in]  stencilWidth  The width of the stencil average rectangle
//! @param[in]  stencilHeight  The height of the stencil average rectangle
//! @param[in]  numLeft  The number of elements to the left of the output element in the stencil pattern
//! @param[in]  numAbove  The number of elements above the output element in the stencil pattern
void stencil(
  const std::vector<float> &srcHost,
  std::vector<float> &dstHost,
  const uint64_t gridWidth,
  const uint64_t gridHeight,
  const uint64_t stencilWidth,
  const uint64_t stencilHeight,
  const uint64_t numLeft,
  const uint64_t numAbove
) {

  assert(!srcHost.empty());
  assert(srcHost.size() == dstHost.size());
  assert(gridWidth * gridHeight == srcHost.size());
  assert(numLeft < stencilWidth);
  assert(numAbove < stencilHeight);

  const uint64_t numBelow = stencilHeight - numAbove - 1;
  const uint64_t numRight = stencilWidth - numLeft - 1;
  const float toDivideBy = static_cast<float>(stencilWidth * stencilHeight);

  hipError_t errorCode;

  float* srcGPU;
	float* dstGPU;
  const size_t gridSz = gridHeight * gridWidth * sizeof(float);
  errorCode = hipMalloc((void **)&srcGPU, gridSz);
  if(errorCode != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << std::endl;
    std::exit(1);
  }
	errorCode = hipMalloc((void **)&dstGPU, gridSz);
  if(errorCode != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << std::endl;
    std::exit(1);
  }

  errorCode = hipMemcpy(srcGPU, srcHost.data(), gridSz, hipMemcpyHostToDevice);
  if(errorCode != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << std::endl;
    std::exit(1);
  }

  dim3 dimBlock(32, 32);
  // Only compute grid cells where the stencil pattern is fully in range
  dim3 dimGrid((gridWidth - stencilWidth + dimBlock.x) / dimBlock.x, (gridHeight - stencilHeight + dimBlock.y) / dimBlock.y);
  
  rectangleStencilAverage<<<dimGrid, dimBlock>>>(
    srcGPU,
    dstGPU,
    toDivideBy,
    gridWidth,
    gridHeight,
    stencilWidth,
    stencilHeight,
    numLeft,
    numRight,
    numAbove,
    numBelow
  );

  errorCode = hipGetLastError();
  if(errorCode != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << std::endl;
    std::exit(1);
  }

  errorCode = hipMemcpy(dstHost.data(), dstGPU, gridSz, hipMemcpyDeviceToHost);
  if(errorCode != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(errorCode) << std::endl;
    std::exit(1);
  }
}

int main(int argc, char* argv[])
{
  struct Params params = getInputParams(argc, argv);

  std::cout << "Running GPU stencil for grid: " << params.gridHeight << "x" << params.gridWidth << std::endl;
  std::cout << "Stencil Size: " << params.stencilHeight << "x" << params.stencilWidth << std::endl;
  std::cout << "Num Above: " << params.numAbove << ", Num Left: " << params.numLeft << std::endl;

  std::vector<float> x, y;
  if (params.inputFile == nullptr)
  {
    // Fill in random grid
    x.resize(params.gridHeight * params.gridWidth);

    #pragma omp parallel
    {
      std::random_device rd;
      std::mt19937 gen(rd());
      std::uniform_real_distribution<float> dist(0.0f, 10000.0f);

      #pragma omp for
      for(size_t i=0; i<x.size(); ++i) {
        x[i] = dist(gen);
      }
    }
  }
  else
  {
    std::cout << "Reading from input file is not implemented yet." << std::endl;
    return 1;
  }

  y.resize(x.size());

  stencil(
    x,
    y,
    params.gridWidth,
    params.gridHeight,
    params.stencilWidth,
    params.stencilHeight,
    params.numLeft,
    params.numAbove
  );

  if (params.shouldVerify)
  {
    bool ok = true;

    // Only compute when stencil is fully in range
    const uint64_t startY = params.numAbove;
    const uint64_t endY = params.gridHeight - (params.stencilHeight - params.numAbove - 1);
    const uint64_t startX = params.numLeft;
    const uint64_t endX = params.gridWidth - (params.stencilWidth - params.numLeft - 1);
    const uint64_t numBelow = params.stencilHeight - params.numAbove - 1;
    const uint64_t numRight = params.stencilWidth - params.numLeft - 1;
    const float toDivideBy = static_cast<float>(params.stencilWidth * params.stencilHeight);

    // CPU and GPU results are not exactly the same
    // TODO: Check if this is okay
    constexpr float acceptableDifference = 0.1f;

    #pragma omp parallel for collapse(2)
    for(uint64_t gridY=startY; gridY<endY; ++gridY) {
      for(uint64_t gridX=startX; gridX<endX; ++gridX) {
        float resCPU = 0.0f;
        for(uint64_t stencilY=gridY-params.numAbove; stencilY<=gridY+numBelow; ++stencilY) {
          for(uint64_t stencilX=gridX-params.numLeft; stencilX<=gridX+numRight; ++stencilX) {
            resCPU += x[stencilY * params.gridWidth + stencilX];
          }
        }
        resCPU /= toDivideBy;
        if (std::abs(resCPU - y[gridY * params.gridWidth + gridX]) > acceptableDifference)
        {
          #pragma omp critical
          {
            std::cout << std::fixed << std::setprecision(3) << "Wrong answer: " << y[gridY * params.gridWidth + gridX] << " (expected " << resCPU << ") at position " << gridX << ", " << gridY << std::endl;
            ok = false;
          }
        }
      }
    }
    if(ok) {
      std::cout << "Correct for stencil!" << std::endl;
    }
  }

  return 0;
}