#include "hip/hip_runtime.h"
/* File:     scale.cu
 * Purpose:  Implement vector scaling on a GPU
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <getopt.h>
#include <assert.h>
#include <stdint.h>
#include <iomanip>
#include <chrono>
#include <omp.h>
#include <iostream>
#include <vector>

#include "utilBaselines.h"

using namespace std;

// Params ---------------------------------------------------------------------
typedef struct Params
{
  uint64_t stringLength;
  uint64_t keyLength;
  char *inputFile;
  bool shouldVerify;
} Params;

void usage()
{
  fprintf(stderr,
          "\nUsage:  ./string-match.out [options]"
          "\n"
          "\n    -s    string size (default=2048 elements)"
          "\n    -k    key size (default = 20 elements)"
          "\n    -c    dramsim config file"
          "\n    -i    input file containing string and key (default=generates strings with random characters)"
          "\n    -v    t = verifies PIM output with host output. (default=false)"
          "\n");
}

struct Params input_params(int argc, char **argv)
{
  struct Params p;
  p.stringLength = 2048;
  p.keyLength = 20;
  p.inputFile = nullptr;
  p.shouldVerify = false;

  int opt;
  while ((opt = getopt(argc, argv, "h:s:k:c:i:v:")) >= 0)
  {
    switch (opt)
    {
    case 'h':
      usage();
      exit(0);
      break;
    case 's':
      p.stringLength = strtoull(optarg, NULL, 0);
      break;
    case 'k':
      p.keyLength = strtoull(optarg, NULL, 0);
      break;
    case 'i':
      p.inputFile = optarg;
      break;
    case 'v':
      p.shouldVerify = (*optarg == 't') ? true : false;
      break;
    default:
      fprintf(stderr, "\nUnrecognized option!\n");
      usage();
      exit(0);
    }
  }
  return p;
}

/**
 * @brief gpu string match kernel
 */
__global__ void string_match(char* haystack, size_t haystack_len, char* needle, size_t needle_len, uint8_t* matches) {
  size_t idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < haystack_len - needle_len + 1) {
    matches[idx] = 1;
    for (int i = 0; i < needle_len; ++i) {
      if (haystack[idx + i] != needle[i]) {
          matches[idx] = 0;
      }
    }
  }
}

void string_match_cpu(string& needle, string& haystack, vector<uint8_t>& matches) {
  size_t pos = haystack.find(needle, 0);

  if (pos == string::npos) {
    return;
  }

  while (pos != string::npos) {
      matches[pos] = 1;
      pos = haystack.find(needle, pos + 1);
  }
}

void getString(string& str, uint64_t len) {
  str.resize(len);
#pragma omp parallel for
  for(uint64_t i=0; i<len; ++i) {
    str[i] = 'a' + (rand()%26);
  }
}

/**
 * @brief Main of the Host Application.
 */
int main(int argc, char **argv)
{
  struct Params params = input_params(argc, argv);
  std::cout << "Running PIM string match for string size: " << params.stringLength << ", key size: " << params.keyLength << "\n";
  string haystack, needle;
  vector<uint8_t> matches;

  if (params.inputFile == nullptr)
  {
    getString(haystack, params.stringLength);
    getString(needle, params.keyLength);
  } 
  else 
  {
    std::cout << "Reading from input file is not implemented yet." << std::endl;
    return 1;
  }

  char* gpu_haystack;
  char* gpu_needle;
  uint8_t* gpu_matches;
  matches.resize(params.stringLength);

  size_t haystack_sz = sizeof(char)*params.stringLength;
  size_t needle_sz = sizeof(char)*params.keyLength;
  size_t matches_sz = sizeof(uint8_t)*params.stringLength;

  hipError_t cuda_error;
  cuda_error = hipMalloc((void**)&gpu_haystack, haystack_sz);

  if(cuda_error != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(cuda_error) << "\n";
    exit(1);
  }

  cuda_error = hipMemcpy(gpu_haystack, haystack.c_str(), haystack_sz, hipMemcpyHostToDevice);

  if(cuda_error != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(cuda_error) << "\n";
    exit(1);
  }

  cuda_error = hipMalloc((void**)&gpu_needle, needle_sz);

  if(cuda_error != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(cuda_error) << "\n";
    exit(1);
  }

  cuda_error = hipMemcpy(gpu_needle, needle.c_str(), needle_sz, hipMemcpyHostToDevice);

  if(cuda_error != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(cuda_error) << "\n";
    exit(1);
  }

  cuda_error = hipMalloc((void**)&gpu_matches, matches_sz);

  if(cuda_error != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(cuda_error) << "\n";
    exit(1);
  }

  cuda_error = hipMemset(gpu_matches, 0, matches_sz);

  if(cuda_error != hipSuccess) {
    std::cerr << "Cuda Error: " << hipGetErrorString(cuda_error) << "\n";
    exit(1);
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float timeElapsed = 0;

  hipEventRecord(start, 0);

  string_match<<<(params.stringLength + 1023) / 1024, 1024>>>(gpu_haystack, params.stringLength, gpu_needle, params.keyLength, gpu_matches);
  
  cuda_error = hipGetLastError();
  if (cuda_error != hipSuccess)
  {
      std::cerr << "Cuda Error: " << hipGetErrorString(cuda_error) << "\n";
      exit(1);
  }

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timeElapsed, start, stop);

  printf("Execution time of string match = %f ms\n", timeElapsed);

  cuda_error = hipMemcpy(matches.data(), gpu_matches, matches_sz, hipMemcpyDeviceToHost);
  if (cuda_error != hipSuccess)
  {
      cerr << "Cuda Error: " << hipGetErrorString(cuda_error) << "\n";
      exit(1);
  }
  hipFree(gpu_haystack);
  hipFree(gpu_needle);
  hipFree(gpu_matches);

  if (params.shouldVerify) 
  {
    vector<uint8_t> matches_cpu;
    matches_cpu.resize(haystack.size());
    string_match_cpu(needle, haystack, matches_cpu);

    // verify result
    // #pragma omp parallel for
    bool is_correct = true;
    for (unsigned i = 0; i < matches.size(); ++i)
    {
      if (matches[i] != matches_cpu[i])
      {
        std::cout << "Wrong answer: " << unsigned(matches[i]) << " (expected " << unsigned(matches_cpu[i]) << "), at index: " << i << std::endl;
        is_correct = false;
      }
    }
    if(is_correct) {
      std::cout << "Correct for string match!" << std::endl;
    }
  }

  return 0;
}
